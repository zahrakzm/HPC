#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

// Simple define to index into a 1D array from 2D space
#define I2D(num, c, r) ((r)*(num)+(c))

/*
 * `step_kernel_mod` is currently a direct copy of the CPU reference solution
 * `step_kernel_ref` below. Accelerate it to run as a CUDA kernel.
 */

__global__ void step_kernel_mod(int ni, int nj, float fact, float* temp_in, float* temp_out)
{
  int i00, im10, ip10, i0m1, i0p1;
  float d2tdx2, d2tdy2;

// define index
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  for ( int index = i; index<(nj-2)*(ni-2);index+=blockDim.x*gridDim.x){

int ix=index%(ni-2)+1;
int jx=index/(nj-2)+1;
//printf("j = %d\n", jx);
//printf("i = %d\n", ix);


i00 = I2D(ni, ix, jx);
im10 = I2D(ni, ix-1, jx);
ip10 = I2D(ni, ix+1, jx);
i0m1 = I2D(ni, ix, jx-1);
i0p1 = I2D(ni, ix, jx+1);

// evaluate derivatives
d2tdx2 = temp_in[im10]-2*temp_in[i00]+temp_in[ip10];
d2tdy2 = temp_in[i0m1]-2*temp_in[i00]+temp_in[i0p1];

// update temperatures
temp_out[i00] = temp_in[i00]+fact*(d2tdx2 + d2tdy2); 
  }
}

void step_kernel_ref(int ni, int nj, float fact, float* temp_in, float* temp_out)
{
  int i00, im10, ip10, i0m1, i0p1;
  float d2tdx2, d2tdy2;


  // loop over all points in domain (except boundary)
  for ( int j=1; j < nj-1; j++ ) {
    for ( int i=1; i < ni-1; i++ ) {
      // find indices into linear memory
      // for central point and neighbours
      i00 = I2D(ni, i, j);
      im10 = I2D(ni, i-1, j);
      ip10 = I2D(ni, i+1, j);
      i0m1 = I2D(ni, i, j-1);
      i0p1 = I2D(ni, i, j+1);

      // evaluate derivatives
      d2tdx2 = temp_in[im10]-2*temp_in[i00]+temp_in[ip10];
      d2tdy2 = temp_in[i0m1]-2*temp_in[i00]+temp_in[i0p1];

      // update temperatures
      temp_out[i00] = temp_in[i00]+fact*(d2tdx2 + d2tdy2);
    }
  }
}

int main(int argc, char* argv[]){
  if(argc != 3){
    printf("Usage: %s <N> <num_threads>\n", argv[0]);
    return 1;
  }

  int istep;
  int nstep = 200; // number of time steps
  int num_threads = atoi(argv[2]);
  hipEvent_t start_malloc, start_gpu, end_gpu, end_malloc;
  float malloc_tot_time, gpu_tot_time;

  // Specify our 2D dimensions
  const int ni = atoi(argv[1]);
  const int nj = atoi(argv[1]);
  float tfac = 8.418e-5; // thermal diffusivity of silver

  float *temp1_ref, *temp2_ref, *temp1, *temp2, *temp_tmp;

  const int size = ni * nj * sizeof(float);

  temp1_ref = (float*)malloc(size);
  temp2_ref = (float*)malloc(size);
  temp1 = (float*)malloc(size);
  temp2 = (float*)malloc(size);

  hipEventCreate(&start_malloc);
  hipEventCreate(&start_gpu);
  hipEventCreate(&end_gpu);
  hipEventCreate(&end_malloc);

  hipEventRecord(start_malloc, 0);
  float *temp1_dev, *temp2_dev, *temp_tmp_dev;
  hipMalloc((void **) &temp1_dev, size);
  hipMalloc((void **) &temp2_dev, size);
  hipMalloc((void **) &temp_tmp_dev, size);

  // Initialize with random data
  for( int i = 0; i < ni*nj; ++i) {
    temp1_ref[i] = temp2_ref[i] = (float)rand()/(float)(RAND_MAX/100.0f);
  }

  hipMemcpy(temp1_dev, temp1_ref, size, hipMemcpyHostToDevice);
  hipMemcpy(temp2_dev, temp2_ref, size, hipMemcpyHostToDevice);

  dim3 threadsPerBlock(num_threads);
  dim3 blocksPerGrid(((ni/2)*(nj/2) + threadsPerBlock.x - 1) / threadsPerBlock.x);

  //clock_t start, end;
  //start = clock();
  // Execute the CPU-only reference version
  for (istep=0; istep < nstep; istep++) {
    step_kernel_ref(ni, nj, tfac, temp1_ref, temp2_ref);

    // swap the temperature pointers
    temp_tmp = temp1_ref;
    temp1_ref = temp2_ref;
    temp2_ref= temp_tmp;
  }
  //end = clock();
  //printf("CPU-only execution time: %f seconds\n", ((double) (end - start)) / CLOCKS_PER_SEC);

  // dim3 threadsPerBlock(num_threads);
  // dim3 blocksPerGrid(((ni/2) * (nj/2) + threadsPerBlock.x - 1) / threadsPerBlock.x);

  hipEventRecord(start_gpu, 0);
  // Execute the modified version using same data
  for (istep=0; istep < nstep; istep++) {
    step_kernel_mod<<<blocksPerGrid, threadsPerBlock>>>(ni, nj, tfac, temp1_dev, temp2_dev);
    hipDeviceSynchronize();
    // swap the temperature pointers
    temp_tmp_dev = temp1_dev;
    temp1_dev = temp2_dev;
    temp2_dev = temp_tmp_dev;
  }
  hipEventRecord(end_gpu, 0);
  hipEventSynchronize(end_gpu);

  hipMemcpy(temp1, temp1_dev, size, hipMemcpyDeviceToHost);
  hipMemcpy(temp2, temp2_dev, size, hipMemcpyDeviceToHost);
  hipEventRecord(end_malloc, 0);
  hipEventSynchronize(end_malloc);

  hipEventElapsedTime(&malloc_tot_time, start_malloc, end_malloc);
  hipEventElapsedTime(&gpu_tot_time, start_gpu, end_gpu);
  printf("GPU execution time: %f seconds\n", gpu_tot_time*1000);
  printf("GPU + Memory allocation execution time: %f seconds\n", malloc_tot_time*1000);

  float maxError = 0;
  // Output should always be stored in the temp1_host and temp1_ref at this point
  for( int i = 0; i < ni*nj; ++i ) {
    if (abs(temp1_host[i]-temp1_ref[i]) > maxError) { maxError = abs(temp1_host[i]-temp1_ref[i]); }
  }

  // Check and see if our maxError is greater than an error bound
  if (maxError > 0.0005f)
    printf("Problem! The Max Error of %.5f is NOT within acceptable bounds.\n", maxError);
  else
    printf("The Max Error of %.5f is within acceptable bounds.\n", maxError);

  hipFree(temp1);
  hipFree(temp2);
  hipFree(temp_tmp);
  free( temp1_ref );
  free( temp2_ref );
  free( temp1 );
  free( temp2 );

  return 0;
}
