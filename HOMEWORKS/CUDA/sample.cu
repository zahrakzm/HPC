#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

// Simple define to index into a 1D array from 2D space
#define I2D(num, c, r) ((r)*(num)+(c))

/*
 * `step_kernel_mod` is currently a direct copy of the CPU reference solution
 * `step_kernel_ref` below. Accelerate it to run as a CUDA kernel.
 */

__global__ void step_kernel_mod(int ni, int nj, float fact, float* temp_in, float* temp_out, int blocksPerGrid)
{
  int i00, im10, ip10, i0m1, i0p1;
  float d2tdx2, d2tdy2;

  int indexWithinTheGrid = blockIdx.x * blockDim.x + threadIdx.x;
  int gridStride = gridDim.x * blockDim.x;
  int N = (ni - 2) * (nj - 2);

  // Calculate the maximum valid index
  int maxIdx = (blocksPerGrid * blockDim.x) - 1;

  for(int k = indexWithinTheGrid; k < maxIdx && k < N; k += gridStride){
    int i = k % (ni - 2) + 1;
    int j = k / (nj - 2) + 1;

    i00 = I2D(ni, i, j);
    im10 = I2D(ni, i - 1, j);
    ip10 = I2D(ni, i + 1, j);
    i0m1 = I2D(ni, i, j - 1);
    i0p1 = I2D(ni, i, j + 1);
    
    // evaluate derivatives
    d2tdx2 = temp_in[im10] - 2 * temp_in[i00] + temp_in[ip10];
    d2tdy2 = temp_in[i0m1] - 2 * temp_in[i00] + temp_in[i0p1];
  
    // update temperatures
    temp_out[i00] = temp_in[i00] + fact * (d2tdx2 + d2tdy2);
  }
}

void step_kernel_ref(int ni, int nj, float fact, float* temp_in, float* temp_out)
{
  int i00, im10, ip10, i0m1, i0p1;
  float d2tdx2, d2tdy2;


  // loop over all points in domain (except boundary)
  for ( int j = 1; j < nj - 1; j++ ) {
    for ( int i = 1; i < ni - 1; i++ ) {
      // find indices into linear memory
      // for central point and neighbours
      i00 = I2D(ni, i, j);
      im10 = I2D(ni, i - 1, j);
      ip10 = I2D(ni, i + 1, j);
      i0m1 = I2D(ni, i, j - 1);
      i0p1 = I2D(ni, i, j + 1);

      // evaluate derivatives
      d2tdx2 = temp_in[im10] - 2 * temp_in[i00] + temp_in[ip10];
      d2tdy2 = temp_in[i0m1] - 2 * temp_in[i00] + temp_in[i0p1];

      // update temperatures
      temp_out[i00] = temp_in[i00] + fact * (d2tdx2 + d2tdy2);
    }
  }
}

int main(int argc, char* argv[]){
  if(argc != 3){
    printf("Usage: %s <N> <num_threads>\n", argv[0]);
    return 1;
  }

  int istep;
  int nstep = 200; // number of time steps
  int num_threads = atoi(argv[2]);

  // Specify our 2D dimensions
  const int ni = atoi(argv[1]);
  const int nj = atoi(argv[1]);
  float tfac = 8.418e-5; // thermal diffusivity of silver

  int threadsPerBlock = num_threads;
  // Calculate the total number of valid elements (excluding boundary)
  int totalElements = (ni - 2) * (nj - 2);

  // Calculate the number of blocks needed
  int blocksPerGrid = (totalElements + threadsPerBlock - 1) / threadsPerBlock;

  float *temp1_ref, *temp2_ref, *temp1_init, *temp2_init;

  const int size = ni * nj * sizeof(float);

  temp1_ref = (float*)malloc(size);
  temp2_ref = (float*)malloc(size);
  temp1_init = (float*)malloc(size);
  temp2_init = (float*)malloc(size);

  // Initialize with random data
  for( int i = 0; i < ni * nj; ++i) {
    temp1_ref[i] = temp2_ref[i] = temp1_init[i] = temp2_init[i] = (float)rand() / (float)(RAND_MAX / 100.0f);
  }

  // CPU-only reference version
  clock_t start, end;
  start = clock();
  for (istep = 0; istep < nstep; istep++) {
    step_kernel_ref(ni, nj, tfac, temp1_ref, temp2_ref);
    // swap the temperature pointers
    float* temp_tmp_ref = temp1_ref;
    temp1_ref = temp2_ref;
    temp2_ref = temp_tmp_ref;
  }
  end = clock();
  printf("CPU-only execution time: %f seconds\n", ((double)(end - start)) / CLOCKS_PER_SEC);

  // GPU version
  hipEvent_t start_malloc, start_gpu, end_gpu, end_malloc;
  hipEventCreate(&start_malloc);
  hipEventCreate(&start_gpu);
  hipEventCreate(&end_gpu);
  hipEventCreate(&end_malloc);

  hipEventRecord(start_malloc, 0);
  float *temp1_dev, *temp2_dev;
  hipMalloc((void **)&temp1_dev, size);
  hipMalloc((void **)&temp2_dev, size);

  hipMemcpy(temp1_dev, temp1_init, size, hipMemcpyHostToDevice);
  hipMemcpy(temp2_dev, temp2_init, size, hipMemcpyHostToDevice);

  hipEventRecord(start_gpu, 0);
  // Execute
  // Execute the modified version using the same data
  for (istep = 0; istep < nstep; istep++) {
    step_kernel_mod<<<blocksPerGrid, threadsPerBlock>>>(ni, nj, tfac, temp1_dev, temp2_dev, blocksPerGrid);
    hipDeviceSynchronize();
    // swap the temperature pointers
    float *temp_tmp = temp1_dev;
    temp1_dev = temp2_dev;
    temp2_dev = temp_tmp;
  }
  hipEventRecord(end_gpu, 0);
  hipEventSynchronize(end_gpu);

  hipMemcpy(temp1_init, temp1_dev, size, hipMemcpyDeviceToHost);
  hipMemcpy(temp2_init, temp2_dev, size, hipMemcpyDeviceToHost);
  hipEventRecord(end_malloc, 0);
  hipEventSynchronize(end_malloc);

  float malloc_tot_time, gpu_tot_time;
  hipEventElapsedTime(&malloc_tot_time, start_malloc, end_malloc);
  hipEventElapsedTime(&gpu_tot_time, start_gpu, end_gpu);
  printf("GPU execution time: %f seconds\n", gpu_tot_time * 1000);
  printf("GPU + Memory allocation execution time: %f seconds\n", malloc_tot_time * 1000);

  hipFree(temp1_dev);
  hipFree(temp2_dev);

  float maxError = 0;
  // Output should always be stored in the temp1 and temp1_ref at this point
  for (int i = 0; i < ni * nj; ++i) {
    if (abs(temp1_init[i] - temp1_ref[i]) > maxError) {
      maxError = abs(temp1_init[i] - temp1_ref[i]);
    }
  }

  // Check and see if our maxError is greater than an error bound
  if (maxError > 0.0005f)
    printf("Problem! The Max Error of %.5f is NOT within acceptable bounds.\n", maxError);
  else
    printf("The Max Error of %.5f is within acceptable bounds.\n", maxError);

  free(temp1_ref);
  free(temp2_ref);
  free(temp1_init);
  free(temp2_init);

  return 0;
}
